#include "hip/hip_runtime.h"
#include "defines.h"

extern "C" __global__ void IncVect(float *Tin, float *Tout) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if(x < N)
		Tout[x] = Tin[x] + 1;
}

