#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

#define		N		1000
#define		BLOCK_SIZE	16

float		HostVect[N];
float	       *DevVectIn, *DevVectOut;
int		blocks;

__global__ void IncVect(float *Tin, float *Tout) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if(x < N)
		Tout[x] = Tin[x] + 1;
}

int main(int argc, char *argv[]) {

	for(int i = 0; i < N; i++)
		HostVect[i] = i;
	hipMalloc((void **) &DevVectIn, sizeof(HostVect));
	hipMalloc((void **) &DevVectOut, sizeof(HostVect));
	hipMemcpy(DevVectIn, HostVect, sizeof(HostVect), hipMemcpyHostToDevice);
	blocks = N / BLOCK_SIZE;
	if(N % BLOCK_SIZE) blocks++;
	IncVect<<<blocks, BLOCK_SIZE>>>(DevVectIn, DevVectOut);
	hipDeviceSynchronize();
	hipMemcpy(HostVect, DevVectOut, sizeof(HostVect), hipMemcpyDeviceToHost);
	hipFree(DevVectIn);
	hipFree(DevVectOut);
	for(int i = 0; i < N; i++)
		assert(HostVect[i] == i + 1);
	puts("done");
	return 0;
}

