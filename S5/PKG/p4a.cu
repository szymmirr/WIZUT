#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <time.h>
using namespace std;

#define  N   		1048576
#define  BLOCK_SIZE	1024

unsigned char 	   hArray[N];
unsigned char     *dArray;
float			hArray2[N];
float			dArray2[N];
long long int 	   blocks;

void epilogue(void) {
	hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
	hipFree(dArray);
}

// Kernel
__global__ void pow3(unsigned char *A) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;

    //if(x < N)
	    //A[x] = A[x] * A[x] * A[x] + A[x] * A[x] + A[x];

	double color1 = A[x];
	double color2 = A[x+1];
	double color3 = A[x+2];
	if(x < N)
	{
    	A[x]=(color1+color2+color3)/3;
    	//A[x+1]=(color1+color2+color3)/3;
    	//A[x+2]=(color1+color2+color3)/3;
		//printf("%f/n",A[x]);
	}
}

struct unpacked {
        char c;
        long l;
};

struct packed {
        char c;
        long l;
} __attribute__ ((packed));


unsigned char* readBMP(char* filename)
{
	clock_t begin, end,begin2,end2;
	double time_spent;
    int i;
    FILE* f = fopen(filename, "rb");
    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];

    int size = 3 * width * height;
    unsigned char* data = new unsigned char[size]; // allocate 3 bytes per pixel
    fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
	unsigned char* data2 = new unsigned char[size]; // allocate 3 bytes per pixel
    fread(data2, sizeof(unsigned char), size, f); // read the rest of the data at once
    fclose(f);

	memset(hArray, 0, sizeof(hArray));
	for(long long int i = 0; i < N; i++) {
		hArray[i] =  data[i];
	}

begin = clock();
    for(i = 0; i < size; i ++)
    {
		double color1 = data[i];
		double color2 = data[i+1];
		double color3 = data[i+2];
        //data[i]=(color1*0.299)+(color2*0.587)+(color3*0.114);
        //data[i+1]=(color1*0.299)+(color2*0.587)+(color3*0.114);
        //data[i+2]=(color1*0.299)+(color2*0.587)+(color3*0.114);
		data[i]=(color1+color2+color3)/3;
        //data[i+1]=(color1+color2+color3)/3;
        //data[i+2]=(color1+color2+color3)/3;
    }
end = clock();
time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
cout <<"CPU time = "<<time_spent << "s"<<endl;

///////////////////////////////////////////////////////////
	int	 devCnt;

	hipGetDeviceCount(&devCnt);
	if(devCnt == 0) {
	perror("No CUDA devices available -- exiting.");
	exit(0);
	}

	
   	hipMalloc((void**)&dArray, sizeof(hArray));
   	hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);

	blocks = N / BLOCK_SIZE;
	if(N % BLOCK_SIZE)
	blocks++;
begin = clock();
	pow3<<<blocks, BLOCK_SIZE>>>(dArray);
end = clock();
time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
cout <<"GPU time = "<<time_spent << "s"<<endl;
	hipDeviceSynchronize();
	
	epilogue();
///////////////////////////////////////////////////////////
    ofstream outfile("malpa2.bmp",ofstream::binary);
    for(i = 0; i<54;i++)
    {
        outfile<<info[i];
    }
    for(i = 0; i<size;i++)
    {
		outfile<<data[i];
    }
    outfile.close();

	ofstream outfile2("malpa3.bmp",ofstream::binary);
    for(i = 0; i<54;i++)
    {
        outfile2<<info[i];
    }
    for(i = 0; i<size;i++)
    {
		outfile2<<hArray[i];
    }
    outfile2.close();
    return data;
}

int main(void)
{
	cout<<fixed;
    //printf("unpacked = %ld\n", sizeof(struct unpacked));
    //printf("packed   = %ld\n", sizeof(struct packed));
	unsigned char* cos = readBMP("malpa.bmp");

    return 0;
}
