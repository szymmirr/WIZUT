#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
using namespace std;

// definiujemy rozmiar problemu i rozmiar bloku
#define NUMBER_OF_ARRAY_ELEMENTS 32
#define N_THREADS_PER_BLOCK 32
// funkcja powiększa o jeden każdy element tablicy a przechowywanej w
// pamięci hosta
void incrementArrayOnHost(float *a, int N)
{
 int i;
 for (i=0; i < N; i++)
a[i] = a[i]+1.f;
}
// kernel powiększa o jeden element tablicy przypadający na
// wykonujący go wątek
__global__ void incrementArrayOnDevice(float *a, int N)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if (idx < N)
a[idx] = a[idx]+1.f;
}

// funkcja pomocnicza (dla miłośników programowania defensywnego)
void checkCUDAError(const char *msg)
{
 hipError_t err = hipGetLastError();
 if( hipSuccess != err) {
 fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
 exit(EXIT_FAILURE);
 }
}

#define BLOCK_SIZE 64
//78 max
// Macierze są pamiętane wierszami, a więc:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
int width;
int height;
int stride;
float *elements;
} Matrix;
// funkcja do odczytywania wartości elementu wskazanej macierzy
__device__ float GetElement(const Matrix A, int row, int col)
{
return A.elements[row * A.stride + col];
}
// funkcja do zapisywania wartości elementu wskazanej macierzy
__device__ void SetElement(Matrix A, int row, int col, float value)
{
A.elements[row * A.stride + col] = value;
}

// wykreowanie opisu podmacierzy o rozmiarze BLOCK_SIZExBLOCK_SIZE, która
// ulokowana jest col podmacierzy w prawo i row podmacierzy w dół
// licząc od lewego wierzchołka danej macierzy
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
Matrix Asub;
Asub.width = BLOCK_SIZE;
Asub.height = BLOCK_SIZE;
Asub.stride = A.stride;
Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row+ BLOCK_SIZE * col];
return Asub;
}


__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
Matrix d_A;
d_A.width = d_A.stride = A.width;
d_A.height = A.height;
size_t size = A.width * A.height * sizeof(float);
hipMalloc((void **)&d_A.elements, size);
hipMemcpy(d_A.elements, A.elements, size,hipMemcpyHostToDevice);
Matrix d_B;
d_B.width = d_B.stride = B.width;
d_B.height = B.height;
size = B.width * B.height * sizeof(float);
hipMalloc((void **)&d_B.elements, size);
hipMemcpy(d_B.elements, B.elements, size,hipMemcpyHostToDevice);
Matrix d_C;
d_C.width = d_C.stride = C.width; d_C.height = C.height;
size = C.width * C.height * sizeof(float);
hipMalloc((void **)&d_C.elements, size);
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
clock_t begin,end;
double time_spent;
begin = clock();


MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

end = clock();
time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
cout<<time_spent<<" czas GPU"<<endl;

hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

hipFree(d_A.elements);
hipFree(d_B.elements);
hipFree(d_C.elements);
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
// ustalenie numeru wiersza i kolumny wewnątrz bloku
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;
// każdy blok oblicza jedną podmacierz Csub macierzy C
Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
// każdy wątek oblicza jeden element Csub akumulując wynik w Cvalue
float Cvalue = 0;
// ustalenie numeru wiersza i kolumny wewnątrz wątku
int row = threadIdx.y;
int col = threadIdx.x;
// iterujemy wszystkie podmacierze A i B, które
// są potrzebne do obliczenia Csub – mnożymy ze sobą każdą parę
// podmacierzy i akumulujemy wynik
for(int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
// kreujemy podmacierz Asub macierzy A
Matrix Asub = GetSubMatrix(A, blockRow, m);
// kreujemy podmacierz Bsub macierzy B
Matrix Bsub = GetSubMatrix(B, m, blockCol);
// deklarujemy obszar pamięci dzielonej dla podmacierzy Asub i Bsub
__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
// załaduj Asub i Bsub z pamięci globalnej do dzielonej
// (każdy wątek ładuje jeden element z każdej podmacierzy)
As[row][col] = GetElement(Asub, row, col);
Bs[row][col] = GetElement(Bsub, row, col);
// poczekajmy, aż wszystkie dane zostaną skopiowane
__syncthreads();
// mnożymy Asub i Bsub
for (int e = 0; e < BLOCK_SIZE; ++e)
Cvalue += As[row][e] * Bs[e][col];
// poczekajmy, aż obliczenia zostaną zakończone zanim zabierzemy
// się za przetwarzanie następnej podmacierzy
__syncthreads();
}
// odsyłamy obliczone Cvalue do pamięci urządzenia
SetElement(Csub, row, col, Cvalue);
}

__global__ void MatMulKernel2(float *A, float *B, float *C)
{
// ustalenie numeru wiersza i kolumny wewnątrz bloku
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;
// każdy blok oblicza jedną podmacierz Csub macierzy C
float Csub = GetSubMatrix(C, blockRow, blockCol);
// każdy wątek oblicza jeden element Csub akumulując wynik w Cvalue
float Cvalue = 0;
// ustalenie numeru wiersza i kolumny wewnątrz wątku
int row = threadIdx.y;
int col = threadIdx.x;
// iterujemy wszystkie podmacierze A i B, które
// są potrzebne do obliczenia Csub – mnożymy ze sobą każdą parę
// podmacierzy i akumulujemy wynik
for(int m = 0; m < ((NUMBER_OF_ARRAY_ELEMENTS/2) / BLOCK_SIZE); ++m) {
// kreujemy podmacierz Asub macierzy A
float Asub = GetSubMatrix(A, blockRow, m);
// kreujemy podmacierz Bsub macierzy B
float Bsub = GetSubMatrix(B, m, blockCol);
// deklarujemy obszar pamięci dzielonej dla podmacierzy Asub i Bsub
__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
// załaduj Asub i Bsub z pamięci globalnej do dzielonej
// (każdy wątek ładuje jeden element z każdej podmacierzy)
As[row][col] = GetElement(Asub, row, col);
Bs[row][col] = GetElement(Bsub, row, col);
// poczekajmy, aż wszystkie dane zostaną skopiowane
__syncthreads();
// mnożymy Asub i Bsub
for (int e = 0; e < BLOCK_SIZE; ++e)
Cvalue += As[row][e] * Bs[e][col];
// poczekajmy, aż obliczenia zostaną zakończone zanim zabierzemy
// się za przetwarzanie następnej podmacierzy
__syncthreads();
}
// odsyłamy obliczone Cvalue do pamięci urządzenia
SetElement(Csub, row, col, Cvalue);
}

int main(int argc, char *argv[])
{
	int mode=0;
	char teges[40]={};
	if(argc<2)
	{
		cout<<"nie podano argumentu: --copy, --map lub --auto"<<endl;
		return 0;
	}
strcpy(teges, argv[1]);
  if(strcmp (teges, "--copy")==0)
  {
    cout<<"copy"<<endl;
	mode=1;
  }
  else if(strcmp (teges, "--map")==0)
  {
    cout<<"map"<<endl;
	mode=2;
  }
  else if(strcmp (teges, "--auto")==0)
  {
    cout<<"auto"<<endl;
	mode=3;
  }
  else
  {
    cout<<"nie podano argumentu: --copy, --map lub --auto"<<endl;
	return 0;
  }
	cout<<fixed;
	if (mode==3)
	{
		mode=2;
		#if CUDART_VERSION < 2020
		#error "To urzadzenie nie wspiera mapowania pamieci ;(\n"
		mode=1;
		#endif
	}
	if (mode==1)
	{
		Matrix A;
		A.width = 40000;
		A.height = 40000;
		A.elements = new float[A.width*A.height];
		Matrix B;
		B.width = 40000;
		B.height = 40000;
		B.elements = new float[B.width*B.height];

		for (int i=0; i< A.width*A.height; i++)
		{
			A.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}

		for (int i=0; i< B.width*B.height; i++)
		{
			B.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}

		Matrix C;
		C.width = 40000;
		C.height = 40000;
		 C.elements = new float[C.width*C.height];
		for (int i=0; i< C.width*C.height; i++)
		{
			C.elements[i] = 0;
		}

		MatMul (A, B, C);
		delete[] A.elements;
		delete[] B.elements;
		delete[] C.elements;
	}

	if (mode==2)
	{
		float *a_m_A; // wskaźnik do pamięci hosta
		 float *a_d_A; // wskaźnik do zamapowanej pamięci urządzenia
		float *a_m_B; // wskaźnik do pamięci hosta
		 float *a_d_B; // wskaźnik do zamapowanej pamięci urządzenia
		float *a_m_C; // wskaźnik do pamięci hosta
		 float *a_d_C; // wskaźnik do zamapowanej pamięci urządzenia
		 int N = NUMBER_OF_ARRAY_ELEMENTS;
		size_t size = NUMBER_OF_ARRAY_ELEMENTS * sizeof(float);
		 hipDeviceProp_t deviceProp;
		#if CUDART_VERSION < 2020
		#error "To urzadzenie nie wspiera mapowania pamieci ;(\n"
		mode=1;
		#endif
		 // Pobierz własności i sprawdź, czy urządzenie #0 wspiera mapowanie
		 hipGetDeviceProperties(&deviceProp, 0);
		 checkCUDAError("hipGetDeviceProperties");
		 if(!deviceProp.canMapHostMemory) {
		 fprintf(stderr, "Urzadzenie %d nie wspiera mapowania pamieci ;(\n", 0);
		 exit(EXIT_FAILURE);
		}
			// przygotuj urządzenie do mapowania pamięci
		 hipSetDeviceFlags(hipDeviceMapHost);
		 checkCUDAError("hipSetDeviceFlags");
		 // przydziel pamięć mapowaną
		 hipHostAlloc((void**)&a_m_A, size, hipHostMallocMapped);
		hipHostAlloc((void**)&a_m_B, size, hipHostMallocMapped);
		hipHostAlloc((void**)&a_m_C, size, hipHostMallocMapped);
		 checkCUDAError("hipHostMallocMapped");
		 // pobierz wskaźnik na pamięć dzielona użyteczny dla urządzenia
		 hipHostGetDevicePointer((void**)&a_d_A, (void*)a_m_A, 0);
		hipHostGetDevicePointer((void**)&a_d_B, (void*)a_m_B, 0);
		hipHostGetDevicePointer((void**)&a_d_C, (void*)a_m_C, 0);
		 checkCUDAError("hipHostGetDevicePointer");
		 // inicjacja danych hosta
		 for (int i=0; i<N; i++)
		{
			a_m_A[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}
		for (int i=0; i<N; i++)
		{
			a_m_B[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}
		for (int i=0; i<N; i++)
		{
			a_m_C[i] = 0;
		}

		 // przygotowanie konfiguracji dla dla odpalenia wątków
		 int blockSize = N_THREADS_PER_BLOCK;
		 int nBlocks = N / blockSize + (N % blockSize > 0 ? 1 : 0);
		// odpalenie wątku
		 //incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
		clock_t begin,end;
		double time_spent;
		begin = clock();
		MatMulKernel2 <<< nBlocks, blockSize >>> (a_d_A, a_d_B, a_d_C);
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout<<time_spent<<" czas GPU z mapowaniem"<<endl;
		 checkCUDAError("incrementArrayOnDevice");
		 hipDeviceSynchronize();
		 checkCUDAError("hipDeviceSynchronize");
		for (int i=0; i< NUMBER_OF_ARRAY_ELEMENTS; i++)
		{
			//cout << a_m_C[i] << endl;
		}
		 hipHostFree(a_m_A); // zwalniamy pamięć dzieloną
		hipHostFree(a_m_B); // zwalniamy pamięć dzieloną
		hipHostFree(a_m_C); // zwalniamy pamięć dzieloną
		}
	return 0;
}
