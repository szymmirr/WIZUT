
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
using namespace std;

#define BLOCK_SIZE 78
//78 max
// Macierze są pamiętane wierszami, a więc:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
int width;
int height;
int stride;
float *elements;
} Matrix;
// funkcja do odczytywania wartości elementu wskazanej macierzy
__device__ float GetElement(const Matrix A, int row, int col)
{
return A.elements[row * A.stride + col];
}
// funkcja do zapisywania wartości elementu wskazanej macierzy
__device__ void SetElement(Matrix A, int row, int col, float value)
{
A.elements[row * A.stride + col] = value;
}

// wykreowanie opisu podmacierzy o rozmiarze BLOCK_SIZExBLOCK_SIZE, która
// ulokowana jest col podmacierzy w prawo i row podmacierzy w dół
// licząc od lewego wierzchołka danej macierzy
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
Matrix Asub;
Asub.width = BLOCK_SIZE;
Asub.height = BLOCK_SIZE;
Asub.stride = A.stride;
Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row+ BLOCK_SIZE * col];
return Asub;
}


__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
Matrix d_A;
d_A.width = d_A.stride = A.width;
d_A.height = A.height;
size_t size = A.width * A.height * sizeof(float);
hipMalloc((void **)&d_A.elements, size);
hipMemcpy(d_A.elements, A.elements, size,hipMemcpyHostToDevice);
Matrix d_B;
d_B.width = d_B.stride = B.width;
d_B.height = B.height;
size = B.width * B.height * sizeof(float);
hipMalloc((void **)&d_B.elements, size);
hipMemcpy(d_B.elements, B.elements, size,hipMemcpyHostToDevice);
Matrix d_C;
d_C.width = d_C.stride = C.width; d_C.height = C.height;
size = C.width * C.height * sizeof(float);
hipMalloc((void **)&d_C.elements, size);
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
clock_t begin,end;
double time_spent;
begin = clock();

MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

end = clock();
time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
cout<<time_spent<<" czas GPU"<<endl;

hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

hipFree(d_A.elements);
hipFree(d_B.elements);
hipFree(d_C.elements);
}


__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
// ustalenie numeru wiersza i kolumny wewnątrz bloku
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;
// każdy blok oblicza jedną podmacierz Csub macierzy C
Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
// każdy wątek oblicza jeden element Csub akumulując wynik w Cvalue
float Cvalue = 0;
// ustalenie numeru wiersza i kolumny wewnątrz wątku
int row = threadIdx.y;
int col = threadIdx.x;
// iterujemy wszystkie podmacierze A i B, które
// są potrzebne do obliczenia Csub – mnożymy ze sobą każdą parę
// podmacierzy i akumulujemy wynik
for(int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
// kreujemy podmacierz Asub macierzy A
Matrix Asub = GetSubMatrix(A, blockRow, m);
// kreujemy podmacierz Bsub macierzy B
Matrix Bsub = GetSubMatrix(B, m, blockCol);
// deklarujemy obszar pamięci dzielonej dla podmacierzy Asub i Bsub
__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
// załaduj Asub i Bsub z pamięci globalnej do dzielonej
// (każdy wątek ładuje jeden element z każdej podmacierzy)
As[row][col] = GetElement(Asub, row, col);
Bs[row][col] = GetElement(Bsub, row, col);
// poczekajmy, aż wszystkie dane zostaną skopiowane
__syncthreads();
// mnożymy Asub i Bsub
for (int e = 0; e < BLOCK_SIZE; ++e)
Cvalue += As[row][e] * Bs[e][col];
// poczekajmy, aż obliczenia zostaną zakończone zanim zabierzemy
// się za przetwarzanie następnej podmacierzy
__syncthreads();
}
// odsyłamy obliczone Cvalue do pamięci urządzenia
SetElement(Csub, row, col, Cvalue);
}


int main(int argc, char** argv)
{
	cout<<fixed;
	Matrix A;
	A.width = 42000;
	A.height = 42000;
	A.elements = new float[A.width*A.height];
	Matrix B;
	B.width = 42000;
	B.height = 42000;
	B.elements = new float[B.width*B.height];

	for (int i=0; i< A.width*A.height; i++)
	{
		A.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	}

	for (int i=0; i< B.width*B.height; i++)
	{
		B.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	}

	Matrix C;
	C.width = 42000;
	C.height = 42000;
	C.elements = new float[C.width*C.height];
	for (int i=0; i< C.width*C.height; i++)
	{
		C.elements[i] = 0;
	}

	MatMul (A, B, C);

	delete[] A.elements;
	delete[] B.elements;
	delete[] C.elements;
	return 0;
}
