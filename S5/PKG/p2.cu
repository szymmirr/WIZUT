
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
using namespace std;

// Macierze są pamiętane wierszami, a więc:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
	int width;
	int height;
	float *elements;
} Matrix;

// definiujemy rozmiar bloku wątków:
#define BLOCK_SIZE 78

// prototyp funkcji mnożącej (kernela)
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Zakładamy (dla uproszczenia rozważań), że wymiary macierzy są
// całkowitymi wielokrotnościami wartości BLOCK_SIZE
void cpu(const Matrix A, const Matrix B, Matrix C)
{
	clock_t begin,end;
	double time_spent;
	begin = clock();

	for(int i = 0; i < A.height; i++)
	 for(int j = 0; j < B.width; j++)
	 {
		 float s = 0;
		 for(int k = 0; k < B.height; k++)
		 {
			 	s += A.elements[i * A.width + k] * B.elements[k * A.width + j];
		 }

		 C.elements[i * C.width + j] = s;
	 }
	 end = clock();
 	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
 	cout<<time_spent<<" czas CPU"<<endl;
}

// Funkcja mnożąca
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// kopiujemy macierze A i B to globalnej pamięci urządzenia
	// najpierw A
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void **)&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	// potem B
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void **)&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size,hipMemcpyHostToDevice);

	// przydzielamy macierz C w globalnej pamięci urządzenia
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void **)&d_C.elements, size);
	// preparujemy środowisko i wywołujemy kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	//czas
	clock_t begin,end;
	double time_spent;
	begin = clock();

	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout<<time_spent<<" czas GPU"<<endl;

	// odbieramy obliczoną macierz C z pamięci globalnej urządzenia
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	// zwalniamy pamięć
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// kernel odpowiedzialny za wymnożenie macierzy
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// każdy wątek oblicza jeden element macierzy C
	// akumulując wynik w zmiennej Cvalue
	float Cvalue = 0;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x +threadIdx.x;

	for (int e = 0; e < A.width; ++e)
	{
		Cvalue += A.elements[row * A.width + e]* B.elements[e * B.width + col];
	}
	C.elements[row * C.width + col] = Cvalue;
}

int main(int argc, char** argv)
{
	cout<<fixed;
	Matrix A;
	A.width = 42000;
	A.height = 42000;
	A.elements = new float[A.width*A.height];
	Matrix B;
	B.width = 42000;
	B.height = 42000;
	B.elements = new float[B.width*B.height];

	for (int i=0; i< A.width*A.height; i++)
	{
		A.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	}

	for (int i=0; i< B.width*B.height; i++)
	{
		B.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	}

	Matrix C;
	C.width = 42000;
	C.height = 42000;
	C.elements = new float[C.width*C.height];
	for (int i=0; i< C.width*C.height; i++)
	{
		C.elements[i] = 0;
	}

	MatMul (A, B, C);

	for (int i=0; i< C.width*C.height; i++)
	{
		C.elements[i] = 0;
	}
	cpu (A,B,C);
	delete[] A.elements;
	delete[] B.elements;
	delete[] C.elements;
	return 0;
}
