
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <iomanip>
#include <cmath>
using namespace std;
#define  N   		1000
#define  BLOCK_SIZE	16
#define  eps	0.0005
float 	   hArray[N];
float     *dArray;
float			hArray2[N];
float			dArray2[N];
int 	   blocks;


void prologue(void) {
	memset(hArray, 0, sizeof(hArray));
	for(int i = 0; i < N; i++) {
		hArray[i] =  i + 1;
	}
   	hipMalloc((void**)&dArray, sizeof(hArray));
   	hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);
}

void cpu()
{
	double n = 2;
    double x = 1;
    float Answer = 1;
	int eps_worked = 0;

    for ( ; n <= 10000; n += 2 )
    {
        Answer = Answer * ( (n*n) / ( x * (x + 2) ) );
        x += 2;
				if(M_PI/2 -Answer < eps)
				{
					cout<<"eps worked"<<endl;
					eps_worked = 1;
					break;
				}
    }
	if (eps_worked == 0)
	{
		cout<<"Reached iteration limit"<<endl;
	}
    cout << "The approximated value for pi/2 = " << Answer << endl;
    cout << "The theoretical value for pi/2 = " << M_PI/2 << endl;
}

void epilogue(void) {
	hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
	hipFree(dArray);
}


// Kernel
__global__ void pow3(float *A) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < N)
	    A[x] = A[x] * A[x] * A[x] + A[x] * A[x] + A[x];
}

int main(int argc, char** argv)
{
		clock_t begin, end,begin2,end2;
		double time_spent;
		begin = clock();
		begin2 = clock();
		cout<<fixed;
		cout<<setprecision(6);
/*
		int	 devCnt;

		cudaGetDeviceCount(&devCnt);
		if(devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
		}

		prologue();
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

		cout<<"Transfer to device = " <<time_spent<<"s"<<endl;

		blocks = N / BLOCK_SIZE;
		if(N % BLOCK_SIZE)
		blocks++;
		pow3<<<blocks, BLOCK_SIZE>>>(dArray);
		//cout << "blocks = " <<blocks<<endl;
		cudaThreadSynchronize();

		begin = clock();
		epilogue();
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout<<"Transfer from device = " <<time_spent<<"s"<<endl;
*/

end2 = clock();
time_spent = (double)(end2 - begin2) / CLOCKS_PER_SEC;
		cout <<"GPU time = "<<time_spent << "s"<<endl;
begin = clock();
cpu();
end = clock();
time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

		cout << "CPU time = " <<time_spent<<"s"<<endl;
    return 0;
}

