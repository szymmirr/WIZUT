#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
using namespace std;

// definiujemy rozmiar problemu i rozmiar bloku
#define NUMBER_OF_ARRAY_ELEMENTS 32
#define N_THREADS_PER_BLOCK 32
// funkcja powiększa o jeden każdy element tablicy a przechowywanej w
// pamięci hosta
void incrementArrayOnHost(float *a, int N)
{
 int i;
 for (i=0; i < N; i++)
a[i] = a[i]+1.f;
}
// kernel powiększa o jeden element tablicy przypadający na
// wykonujący go wątek
__global__ void incrementArrayOnDevice(float *a, int N)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if (idx < N)
a[idx] = a[idx]+1.f;
}

// funkcja pomocnicza (dla miłośników programowania defensywnego)
void checkCUDAError(const char *msg)
{
 hipError_t err = hipGetLastError();
 if( hipSuccess != err) {
 fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
 exit(EXIT_FAILURE);
 }
}

// Macierze są pamiętane wierszami, a więc:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
	int width;
	int height;
	float *elements;
} Matrix;

// definiujemy rozmiar bloku wątków:
#define BLOCK_SIZE 3200

// prototyp funkcji mnożącej (kernela)
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


// Zakładamy (dla uproszczenia rozważań), że wymiary macierzy są
// całkowitymi wielokrotnościami wartości BLOCK_SIZE
void cpu(const Matrix A, const Matrix B, Matrix C)
{
	clock_t begin,end;
	double time_spent;
	begin = clock();
	for(int i = 0; i < A.height; i++)
	 for(int j = 0; j < B.width; j++)
	 {
		 float s = 0;
		 for(int k = 0; k < B.height; k++)
		 {
			 	s += A.elements[i * A.width + k] * B.elements[k * A.width + j];
		 }
		 C.elements[i * C.width + j] = s;
	 }
	 end = clock();
 	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
 	cout<<time_spent<<" czas CPU"<<endl;
	 cout<<"wynik cpu"<<endl;
	 for (int i=0; i< C.width*C.height; i++)
 	{
 		//cout << C.elements[i] << endl;
 	}
}
// Funkcja mnożąca
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// kopiujemy macierze A i B to globalnej pamięci urządzenia
	// najpierw A
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void **)&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	// potem B
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void **)&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size,hipMemcpyHostToDevice);

	// przydzielamy macierz C w globalnej pamięci urządzenia
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void **)&d_C.elements, size);
	// preparujemy środowisko i wywołujemy kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	//czas
	clock_t begin,end;
	double time_spent;
	begin = clock();

	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout<<time_spent<<" czas GPU"<<endl;

	// odbieramy obliczoną macierz C z pamięci globalnej urządzenia
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	// zwalniamy pamięć
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// kernel odpowiedzialny za wymnożenie macierzy
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// każdy wątek oblicza jeden element macierzy C
	// akumulując wynik w zmiennej Cvalue
	float Cvalue = 0;
//printf ("cvalue %d \n", Cvalue);
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x +threadIdx.x;
	//cuPrintf("cvalue %d /n", Cvalue);

	for (int e = 0; e < A.width; ++e)
	{
		Cvalue += A.elements[row * A.width + e]* B.elements[e * B.width + col];
	}

//	printf ("cvalue %f \n", Cvalue);
	C.elements[row * C.width + col] = Cvalue;
}

__global__ void MatMulKernel2(float *A, float *B, float *C)
{
	// każdy wątek oblicza jeden element macierzy C
	// akumulując wynik w zmiennej Cvalue
	float Cvalue = 0;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x +threadIdx.x;

	for (int e = 0; e < NUMBER_OF_ARRAY_ELEMENTS/2; ++e)
	{
		Cvalue += A[row * (NUMBER_OF_ARRAY_ELEMENTS/2) + e]* B[e * (NUMBER_OF_ARRAY_ELEMENTS/2) + col];
	}

	//printf ("cvalue %f \n", Cvalue);
	C[row * (NUMBER_OF_ARRAY_ELEMENTS/2) + col] = Cvalue;
}

int main(int argc, char *argv[])
{
	int mode=0;
	char teges[40]={};
	if(argc<2)
	{
		cout<<"nie podano argumentu: --copy, --map lub --auto"<<endl;
		return 0;
	}
strcpy(teges, argv[1]);
  if(strcmp (teges, "--copy")==0)
  {
    cout<<"copy"<<endl;
	mode=1;
  }
  else if(strcmp (teges, "--map")==0)
  {
    cout<<"map"<<endl;
	mode=2;
  }
  else if(strcmp (teges, "--auto")==0)
  {
    cout<<"auto"<<endl;
	mode=3;
  }
  else
  {
    cout<<"nie podano argumentu: --copy, --map lub --auto"<<endl;
	return 0;
  }
	cout<<fixed;
	
	if (mode==3)
	{
		mode=2;
		#if CUDART_VERSION < 2020
		#error "To urzadzenie nie wspiera mapowania pamieci ;(\n"
		mode=1;
		#endif
	}
	if (mode==1)
	{
		Matrix A;
		A.width = 32;
		A.height = 32;
		A.elements = new float[A.width*A.height];
		Matrix B;
		B.width = 32;
		B.height = 32;
		B.elements = new float[B.width*B.height];

		for (int i=0; i< A.width*A.height; i++)
		{
			A.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}

		for (int i=0; i< B.width*B.height; i++)
		{
			B.elements[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}
		Matrix C;
		C.width = 32;
		C.height = 32;
		 C.elements = new float[C.width*C.height];
		for (int i=0; i< C.width*C.height; i++)
		{
			C.elements[i] = 0;
		}
		MatMul (A, B, C);
		delete[] A.elements;
		delete[] B.elements;
		delete[] C.elements;
	}
	if (mode==2)
	{
		float *a_m_A; // wskaźnik do pamięci hosta
		 float *a_d_A; // wskaźnik do zamapowanej pamięci urządzenia
		float *a_m_B; // wskaźnik do pamięci hosta
		 float *a_d_B; // wskaźnik do zamapowanej pamięci urządzenia
		float *a_m_C; // wskaźnik do pamięci hosta
		 float *a_d_C; // wskaźnik do zamapowanej pamięci urządzenia
		 int N = NUMBER_OF_ARRAY_ELEMENTS;
		size_t size = NUMBER_OF_ARRAY_ELEMENTS * sizeof(float);
		 hipDeviceProp_t deviceProp;
		#if CUDART_VERSION < 2020
		#error "To urzadzenie nie wspiera mapowania pamieci ;(\n"
		mode=1;
		#endif
		 // Pobierz własności i sprawdź, czy urządzenie #0 wspiera mapowanie
		 hipGetDeviceProperties(&deviceProp, 0);
		 checkCUDAError("hipGetDeviceProperties");
		 if(!deviceProp.canMapHostMemory) {
		 fprintf(stderr, "Urzadzenie %d nie wspiera mapowania pamieci ;(\n", 0);
		 exit(EXIT_FAILURE);
		}
			// przygotuj urządzenie do mapowania pamięci
		 hipSetDeviceFlags(hipDeviceMapHost);
		 checkCUDAError("hipSetDeviceFlags");
		 // przydziel pamięć mapowaną
		 hipHostAlloc((void**)&a_m_A, size, hipHostMallocMapped);
		hipHostAlloc((void**)&a_m_B, size, hipHostMallocMapped);
		hipHostAlloc((void**)&a_m_C, size, hipHostMallocMapped);
		 checkCUDAError("hipHostMallocMapped");
		 // pobierz wskaźnik na pamięć dzielona użyteczny dla urządzenia
		 hipHostGetDevicePointer((void**)&a_d_A, (void*)a_m_A, 0);
		hipHostGetDevicePointer((void**)&a_d_B, (void*)a_m_B, 0);
		hipHostGetDevicePointer((void**)&a_d_C, (void*)a_m_C, 0);
		 checkCUDAError("hipHostGetDevicePointer");
		 // inicjacja danych hosta
		 for (int i=0; i<N; i++)
		{
			a_m_A[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}
		for (int i=0; i<N; i++)
		{
			a_m_B[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}
		for (int i=0; i<N; i++)
		{
			a_m_C[i] = 0;
		}

		 // przygotowanie konfiguracji dla dla odpalenia wątków
		 int blockSize = N_THREADS_PER_BLOCK;
		 int nBlocks = N / blockSize + (N % blockSize > 0 ? 1 : 0);
		// odpalenie wątku
		 //incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
		clock_t begin,end;
		double time_spent;
		begin = clock();
		MatMulKernel2 <<< nBlocks, blockSize >>> (a_d_A, a_d_B, a_d_C);
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout<<time_spent<<" czas GPU z mapowaniem"<<endl;
		 checkCUDAError("incrementArrayOnDevice");
		 hipDeviceSynchronize();
		 checkCUDAError("hipDeviceSynchronize");
		for (int i=0; i< NUMBER_OF_ARRAY_ELEMENTS; i++)
		{
			//cout << a_m_C[i] << endl;
		}
		 hipHostFree(a_m_A); // zwalniamy pamięć dzieloną
		hipHostFree(a_m_B); // zwalniamy pamięć dzieloną
		hipHostFree(a_m_C); // zwalniamy pamięć dzieloną
		}
	return 0;
}



