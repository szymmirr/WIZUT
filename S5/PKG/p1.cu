#include "hip/hip_runtime.h"

#include <stdio.h>

#define  N   		1000
#define  BLOCK_SIZE	16

float 	   hArray[N];
float     *dArray;
int 	   blocks;


void prologue(void) {
	memset(hArray, 0, sizeof(hArray));
	for(int i = 0; i < N; i++) {
		hArray[i] =  i + 1;
	}		
   	hipMalloc((void**)&dArray, sizeof(hArray));
   	hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);
}

void epilogue(void) {
	hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
	hipFree(dArray);
}


// Kernel
__global__ void pow3(float *A) {
	int x = blockDim.x * blockIdx.x + threadIdx.x; 

    if(x < N)
	    A[x] = A[x] * A[x] * A[x] + A[x] * A[x] + A[x]; 
}

int main(int argc, char** argv)
{
    int	 devCnt;

    hipGetDeviceCount(&devCnt);
    if(devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
    }

    prologue();
    blocks = N / BLOCK_SIZE;
    if(N % BLOCK_SIZE)
		blocks++;
    pow3<<<blocks, BLOCK_SIZE>>>(dArray);
    hipDeviceSynchronize();
    epilogue();
    return 0;
}
