#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <iomanip>
using namespace std;
#define  N   		1000
#define  BLOCK_SIZE	1024

float 	   hArray[N];
float     *dArray;
float			hArray2[N];
float			dArray2[N];
int 	   blocks;


void prologue(void) {
	memset(hArray, 0, sizeof(hArray));
	for(int i = 0; i < N; i++) {
		hArray[i] =  i + 1;
	}
   	hipMalloc((void**)&dArray, sizeof(hArray));
   	hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);
}

void cpu()
{
	memset(hArray2, 0, sizeof(hArray2));
	for(int i = 0; i < N; i++) {
		hArray2[i] =  i + 1;
	}
	for(int i = 0; i < N; i++) {
		dArray2[i] =  0;
	}
	for(int i = 0; i<N;i++)
	{
		dArray2[i] = hArray2[i] * hArray2[i] * hArray2[i] + hArray2[i] * hArray2[i] + hArray2[i];

	}
}

void epilogue(void) {
	hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
	hipFree(dArray);
}


// Kernel
__global__ void pow3(float *A) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < N)
	    A[x] = A[x] * A[x] * A[x] + A[x] * A[x] + A[x];
}

int main(int argc, char** argv)
{
		clock_t begin, end,begin2,end2;
		double time_spent;
		begin = clock();
		begin2 = clock();
		cout<<fixed;
		cout<<setprecision(6);

		int	 devCnt;

		hipGetDeviceCount(&devCnt);
		if(devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
		}

		prologue();
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

		cout<<"Transfer to device = " <<time_spent<<"s"<<endl;

		blocks = N / BLOCK_SIZE;
		if(N % BLOCK_SIZE)
		blocks++;
		pow3<<<blocks, BLOCK_SIZE>>>(dArray);
		hipDeviceSynchronize();

		begin = clock();
		epilogue();
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout<<"Transfer from device = " <<time_spent<<"s"<<endl;


end2 = clock();
time_spent = (double)(end2 - begin2) / CLOCKS_PER_SEC;
		cout <<"GPU time = "<<time_spent << "s"<<endl;
begin = clock();
cpu();
end = clock();
time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

		cout << "CPU time = " <<time_spent<<"s"<<endl;
    return 0;
}

